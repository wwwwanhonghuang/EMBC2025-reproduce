#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <yaml-cpp/yaml.h>
#include <string>
#include <memory>
#include <grammar/grammar_parser.hpp>
#include "utils/grammar_loader.cuh"
#include "device_management.cuh"
#include "macros.def"
#include <stdio.h>

// __global__ void initialize_buffers(){
    
// }
// Kernel function to initialize the CKY buffer

__device__ float logsumexpf(float a, float b);

struct AlgorithmContext{
    int S = 0;
    int MAX_SEQ_LEN = 0;
    cuda_gc_managed_pt<float> CKY;
    cuda_gc_managed_pt<float> grammar;
    cuda_gc_managed_pt<int> sequence;
    cuda_gc_managed_pt<float> intermediate_results_buffer;
    cuda_gc_managed_pt<int> d_changed;
    std::shared_ptr<CudaGC> cuda_gc;

};


void initialize_buffers(AlgorithmContext context){
    context.cuda_gc->fill(context.CKY, -INFINITY);
    context.cuda_gc->zerolize(context.d_changed);
}


YAML::Node read_yaml_configuration(const std::string& configuration_file_path){
    try {
        YAML::Node config = YAML::LoadFile(configuration_file_path);
        return config;
    } catch (const YAML::Exception& e) {
        std::cerr << "Error loading YAML file: " << e.what() << std::endl;
        return YAML::Node();
    }
}


__global__ void cky_initialization_kernel(int S, int MAX_SEQ_LEN,
    float* __restrict__ cky_ptr, float* __restrict__ grammar_ptr,
    int* __restrict__ sequence, int* __restrict__ d_changed) {

    // Terminate cases: A -> word (length-1 spans)
    for (int s_A = blockIdx.y * blockDim.y + threadIdx.y;
        s_A < S;
        s_A += blockDim.y * gridDim.y) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x;
            i < MAX_SEQ_LEN;
            i += blockDim.x * gridDim.x) {

            int word = sequence[i];
            int grammar_idx = s_A * (S + 1) * (S + 1) + word * (S + 1);
            float rule_val = grammar_ptr[grammar_idx];
            
            int cky_idx = s_A * (MAX_SEQ_LEN * MAX_SEQ_LEN) + i * MAX_SEQ_LEN + i;
            cky_ptr[cky_idx] = rule_val;
        }
    }

    __syncthreads();

    // Process unary rules (A->B) with convergence detection
    for (int step = 0; step < S; step++) {
        bool thread_changed = false;
        
        for (int s_A = blockIdx.y * blockDim.y + threadIdx.y;
            s_A < S;
            s_A += blockDim.y * gridDim.y) {
            for (int i = blockIdx.x * blockDim.x + threadIdx.x;
                i < MAX_SEQ_LEN;
                i += blockDim.x * gridDim.x) {

                float current_val = cky_ptr[s_A * (MAX_SEQ_LEN * MAX_SEQ_LEN) + i * MAX_SEQ_LEN + i];
                float max_val = current_val;

                // Check all possible unary rules A->B
                for (int s_B = 0; s_B < S; s_B++) {
                    int grammar_idx = s_A * (S + 1) * (S + 1) + s_B * (S + 1);
                    float rule_val = grammar_ptr[grammar_idx];
                    float b_val = cky_ptr[s_B * (MAX_SEQ_LEN * MAX_SEQ_LEN) + i * MAX_SEQ_LEN + i];
                    float candidate = rule_val + b_val;

                    if (candidate > max_val) {
                        max_val = candidate;
                    }
                }

                if (max_val > current_val) {
                    cky_ptr[s_A * (MAX_SEQ_LEN * MAX_SEQ_LEN) + i * MAX_SEQ_LEN + i] = max_val;
                    thread_changed = true;
                }
            }
        }

        // Efficient convergence check using atomic operation
        if (thread_changed) {
            atomicOr(d_changed, 1);
        }

        __syncthreads();
        
        // Early exit if no changes
        if (step > 0 && !(*d_changed)) {
            break;
        }
        
        // Reset for next iteration
        if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
            *d_changed = 0;
        }
        
        __syncthreads();
    }
}

// Helper function for atomic float max
__device__ void atomicMaxFloat(float* address, float val) {
    int* address_as_int = (int*)address;
    int old = *address_as_int, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_int, assumed,
            __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
}

__global__ void cky_reduce_kernel(
    int S, 
    int MAX_SEQ_LEN,
    float* __restrict__ cky_table,
    float* __restrict__ intermediate_buffer
){

    // Parallelize across 3D grid: s_A, i, j
    int s_A = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.z * blockDim.z + threadIdx.z;
    

    // Boundary checks
    if (s_A >= S || i >= MAX_SEQ_LEN || j >= MAX_SEQ_LEN) 
        return;

    float reduced_val = -INFINITY;
    int base_idx = s_A * S * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j;

    // Each thread reduces across s_B dimension
    for (int s_B = 0; s_B < S; s_B++) {
        int buffer_idx = base_idx + s_B * MAX_SEQ_LEN * MAX_SEQ_LEN;
        reduced_val = logsumexpf(reduced_val, intermediate_buffer[buffer_idx]);
    }

    // Write reduced result to CKY table
    cky_table[s_A * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j] = reduced_val;
}


__device__ float logsumexpf(float a, float b) {
    if (a == -INFINITY) return b;
    if (b == -INFINITY) return a;
    float max_ab = fmaxf(a, b);
    return max_ab + logf(expf(a - max_ab) + expf(b - max_ab));
}

__global__ void cky_span_processing_kernel(
    int span_length, int S, int MAX_SEQ_LEN,
    float* __restrict__ cky,
    float* __restrict__ grammar,
    float* __restrict__ results,
    float* unary_chain, int unary_chain_length)
{
    // Parallelize over spans and non-terminals
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int s_A = blockIdx.y * blockDim.y + threadIdx.y;
    int s_B = blockIdx.z * blockDim.z + threadIdx.z;

    // Boundary checks
    if (i >= MAX_SEQ_LEN - span_length + 1 || s_A > S || s_B > S) return;
    int j = i + span_length - 1;
    if (j >= MAX_SEQ_LEN) return;

    float total_score = -INFINITY;
    const int grammar_stride = (S + 1) * (S + 1);
    float epsilon_rule = grammar[s_A * grammar_stride + s_B * (S + 1) + 0];

    // Process all possible splits
    for (int k = i; k < j; k++) {
        float left_score = cky[s_B * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + k];
        if (left_score == -INFINITY) continue;
        
        // Process binary productions (A -> B C)
        for (int s_C = 1; s_C <= S; s_C++) {  // Skip epsilon (0)
            float right_score = cky[s_C * MAX_SEQ_LEN * MAX_SEQ_LEN + (k + 1) * MAX_SEQ_LEN + j];
            if (right_score == -INFINITY) continue;
            
            float rule = grammar[s_A * grammar_stride + s_B * (S + 1) + s_C];
            total_score = logsumexpf(total_score, left_score + right_score + rule);
        }
    }

    // Handle unary production (A -> B) for this span
    float b_score = cky[s_B * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j];
    if (b_score != -INFINITY) {
        total_score = logsumexpf(total_score, b_score + epsilon_rule);
    }

    // Store result
    int index = s_A * (S + 1) * MAX_SEQ_LEN * MAX_SEQ_LEN
              + s_B * MAX_SEQ_LEN * MAX_SEQ_LEN 
              + i * MAX_SEQ_LEN 
              + j;
    
    // Atomic update to handle potential conflicts
    if (total_score != -INFINITY) {
        printf("set intermediate result [%d, %d, %d, %d] = %lf", s_A, s_B, i, j, total_score);
        atomicMaxFloat(&results[index], total_score);
    }

    __syncthreads();


    // reduce s_B axis
    if(blockIdx.z * blockDim.z + threadIdx.z == 0){
        for(int s = 1; s <= S; s++){
            cky[s_A * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j] =
                logsumexpf(
                    cky[s_A * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j], 
                    results[
                        s_A * (S + 1) * MAX_SEQ_LEN * MAX_SEQ_LEN + 
                        s_B * MAX_SEQ_LEN * MAX_SEQ_LEN + 
                        i * MAX_SEQ_LEN +
                        j]
                );
        }
    }

    __syncthreads();



    // Process unary rules (A -> B)
    if(s_A == 0 && s_B == 0){
        for(int unary_rule_id_in_chain = 0; unary_rule_id_in_chain < unary_chain_length; unary_rule_id_in_chain += 2){
            int unary_rule_s_A = unary_chain[unary_rule_id_in_chain];
            int unary_rule_s_B = unary_chain[unary_rule_id_in_chain + 1];
            cky[unary_rule_s_A * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j]
                = logsumexpf(cky[unary_rule_s_A * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j], 
                    cky[unary_rule_s_B * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j] + grammar[unary_rule_s_A * (S + 1) * (S + 1) + unary_rule_s_B * (S + 1)]
                );
        }
    }
    
}

void cuda_cky_algorithm(AlgorithmContext context) {
    std::cout << "Begin CKY algorithm..." << std::endl;
    std::cout << "Zero out CKY Buffer..." << std::endl;

    initialize_buffers(context);
    std::cout << "[Completed] Zero out CKY Buffer." << std::endl;

    // Launch the kernel to initialize the CKY table
    const int BLOCK_X = 128;  
    const int BLOCK_Y = 8; 

    // Compute grid size (adjust based on your GPU limits)
    int grid_x = min((context.MAX_SEQ_LEN + BLOCK_X - 1) / BLOCK_X, 65535);
    int grid_y = min((context.S + BLOCK_Y - 1) / BLOCK_Y, 65535);

    dim3 blocks(grid_x, grid_y);
    dim3 threads(BLOCK_X, BLOCK_Y);


    std::cout << "Launch CKY span 1 calcualtion kernel..." <<  std::endl;

    cky_initialization_kernel<<<blocks, threads>>>(context.S, context.MAX_SEQ_LEN, 
        context.CKY.ptr, context.grammar.ptr, context.sequence.ptr, context.d_changed.ptr);
    

    hipError_t cudaerr = hipPeekAtLastError();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));

    std::cout << "[Completed] CKY span 1 calcualtion." << std::endl;
    
    /* In the CKY algorithm, tasks with a particular span length represent the largest
       parallelizable units of computation. Therefore, we set the largest grain of
       parallelism to the computation over a specific span length. */
    dim3 cky_blockDim(64, 4, 4);  // Each block has N x S x S threads
    dim3 cky_gridDim((context.MAX_SEQ_LEN + 64 - 1) / 64, (context.S + 4 - 1) / 4, (context.S + 4 - 1) / 4); 
    
    for(int span_length = 2; span_length < context.MAX_SEQ_LEN; span_length++) {
       
        cky_span_processing_kernel<<<cky_gridDim, cky_blockDim>>>(
            span_length, context.S, context.MAX_SEQ_LEN, 
            context.CKY.ptr, context.grammar.ptr, context.intermediate_results_buffer.ptr);
        hipDeviceSynchronize();
       
        break;

    }
    // cky_reduce_kernel<<<cky_gridDim, cky_blockDim>>>(context.S, context.MAX_SEQ_LEN, context.CKY.ptr, context.intermediate_results_buffer.ptr);
    
    std::cout << "[Completed] CKY Algorithm." << std::endl;

}


int main(int argc, char* argv[]) {
    std::string configuration_file_path = "./configurations/config.yaml";
    AlgorithmContext context;

    std::shared_ptr<CudaGC> cuda_gc = std::shared_ptr<CudaGC>();
    context.cuda_gc = cuda_gc;

    if (argc >= 2) {
        configuration_file_path = std::string(argv[1]);
    }

    YAML::Node config = read_yaml_configuration(configuration_file_path);
    if (config.IsNull()) {
        std::cerr << "Failed to load configuration file!" << std::endl;
        return -1;  // Handle the error
    }
    
    int use_device_id = config["cuda_device"]["use_device_id"].as<int>();
    if(select_cuda_device(use_device_id) == 0){
        std::cout << "Use CUDA Device ID: " << use_device_id << std::endl;
    }else{
        return -1;
    }

    int MAX_SEQ_LEN = config["cky_buffer"]["size"]["max_seq_len"].as<int>();
    const std::string& grammar_file_path =  config["grammar"]["file_path"].as<std::string>();
    std::cout << "grammar file path = " << grammar_file_path << std::endl;
    pcfg* parsed_pcfg = prepare_grammar(grammar_file_path);
    __host_pt__ float* host_grammar_buffer = initialize_grammar_buffer_from_pcfg(parsed_pcfg);
    int S = parsed_pcfg->nonterminate_map.size() + parsed_pcfg->terminate_map.size();


    size_t n_cky_buffer_elements = (S + 1) * MAX_SEQ_LEN * MAX_SEQ_LEN;
    size_t n_grammar_buffer_elements = (S + 1) * (S + 1) * (S + 1);
    size_t n_sequence_buffer_elements = MAX_SEQ_LEN; // A -> B C
    long n_intermediate_results_buffer_elements = (S + 1) * (S + 1) * MAX_SEQ_LEN * MAX_SEQ_LEN; // [A, B, i, j]
    std::cout << MAX_SEQ_LEN << "," << S << ", " << S * S * MAX_SEQ_LEN * MAX_SEQ_LEN  << std::endl;

    
    cuda_gc_managed_pt<float> d_CKY = cuda_gc->allocate<float>(n_cky_buffer_elements);
    cuda_gc_managed_pt<float> grammar = cuda_gc->allocate<float>(n_grammar_buffer_elements);
    cuda_gc_managed_pt<int> sequence = cuda_gc->allocate<int>(n_sequence_buffer_elements);
    cuda_gc_managed_pt<int> d_changed = cuda_gc->allocate<int>(1);

    cuda_gc->zerolize(grammar);
    cuda_gc->zerolize(sequence);
    cuda_gc->zerolize(d_changed);

    cuda_gc_managed_pt<float> intermediate_results_buffer = cuda_gc->allocate<float>(n_intermediate_results_buffer_elements);
    context.S = S;
    context.MAX_SEQ_LEN = MAX_SEQ_LEN;
    context.CKY = d_CKY;
    context.intermediate_results_buffer = intermediate_results_buffer;
    context.d_changed = d_changed;
    
    initialize_buffers(context);
    cuda_gc->fill(intermediate_results_buffer, -INFINITY);
    hipDeviceSynchronize();

    auto inside_order_1_rule_iteration_path = generate_inside_perterminate_iteration_paths(parsed_pcfg);
    

    __host_pt__ int* host_sequence = new int[MAX_SEQ_LEN];
    
    /* [fish people fish tanks]'s ID sequence == [10 9 10 11] + 1*/
    host_sequence[0] = 10;
    host_sequence[1] = 9;
    host_sequence[2] = 10;
    host_sequence[3] = 11;
    host_sequence[4] = 0;
    host_sequence[5] = 0;
    host_sequence[6] = 0;
    host_sequence[7] = 0;
    host_sequence[8] = 0;
    host_sequence[9] = 0;

    hipMemcpy(grammar.ptr, host_grammar_buffer, n_grammar_buffer_elements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sequence.ptr, host_sequence, n_sequence_buffer_elements * sizeof(int), hipMemcpyHostToDevice);
    context.sequence = sequence;
    context.grammar = grammar;
    cuda_cky_algorithm(context);

    /* Process data in host. */
    // For demonstration: copy a small part of CKY to the host and print a value
    __host_pt__ float* h_CKY = new float[n_cky_buffer_elements];  // Allocate host memory
    hipMemcpy(h_CKY, d_CKY.ptr, n_cky_buffer_elements * sizeof(float), hipMemcpyDeviceToHost);  // Copy data from device to host

    // Print a value for demonstration (example: CKY[0][0][0])
    
    for(int i = 0; i < 4; i++){
        for(int j = i; j < 4; j++){
            for(int s = 0; s < S; s++){
                std::cout << "CKY[" << s << "][" << i << "][" << j << "]: " 
                << std::exp(h_CKY[s * MAX_SEQ_LEN * MAX_SEQ_LEN + i * MAX_SEQ_LEN + j]) << std::endl;
            }
        }
    }
    

    // Clean up
    delete[] h_CKY; 
    cuda_gc->deallocate<float>(d_CKY);
    hipDeviceReset();

    return 0;
}
